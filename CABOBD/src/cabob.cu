#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "catsparser.cuh"
/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {					\
		hipError_t _m_cudaStat = value;			\
		if (_m_cudaStat != hipSuccess) {			\
			fprintf(stderr, "Error %s at line %d in file %s\n", \
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
					exit(1);					\
		} }

__device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}
template<int _rows,int _collumns>

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}

struct stats {
	int collumns;
	int start;
};

//make sure cost is aligned to 32-256 overspend and nulled to 0.0f
#define DEFAULT_FLOAT (0.0f)
template<int lim>
__device__ int __forceinline__ find_entering_var_old(float * cost, int n) {
	const unsigned int tid = threadIdx.x;
	const unsigned int warpid = tid/32;
	const unsigned int laneid = tid%32;

	float vmax = 0.0f;
	int imax = laneid;
	/*
	 * cache[0]. x y z w
	 *           0 1 2 3
	 * cache[1]. x y z w
	 *           128 129 130 131
	 */
	if(lim <= 32) {
		if(laneid < n) {
			vmax = cost[laneid];
		}
	} else if(lim <= 64) {
		float2 cache = make_float2(DEFAULT_FLOAT,DEFAULT_FLOAT);
		imax = laneid*2;
		if(laneid*2 < n) {
			cache = ((float2 *) cost)[laneid];
			cache.y *= (laneid*2+1 < n);

		}
		vmax = fmax(cache.x,cache.y);
		if(cache.y == vmax) {
			imax++;
		}
	} else if (lim <= 128) {
		float4 cache = make_float4(DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT);

		if(laneid*4 < n) {
			cache = ((float4 *) cost)[laneid];
			cache.y *= (laneid*4+1 < n);
			cache.z *= (laneid*4+2 < n);
			cache.w *= (laneid*4+3 < n);
		}
		cache.x = fmax(cache.x,cache.y);
		cache.z = fmax(cache.z,cache.w);
		vmax = fmax(cache.x,cache.z);
		//set base index to x
		imax = laneid*4;
		if (vmax == cache.y) {
			imax +=1;
		} else if(vmax == cache.z) {
			imax += 2;
		} else if (vmax == cache.w) {
			imax += 3;
		}
	} else if ( lim <= 256) {

		float4 cache[2];
		cache[0] = ((float4 *) cost)[laneid];//guaranteed to be in mem, as n > 128
		cache[1] = ((float4 *) cost)[laneid+32];
		if(laneid*4+4+4*32 >= n) {
			// ignore constant 4*32 in calc
			/* laneid 0: n 1: 0+4-1= 3: yzw
			 * laneid 0: n 2: 0+4-2= 2: zw
			 * laneid 0: n 3: 0+4-3= 1: w
			 * laneid 0: n 4: 0+4-4= 0:
			 * laneid 4: n 4: 16+4-4= 16: xyzw
			 * laneid 3: n 4: 3+4-4= 3: yzw
			 */
			switch(laneid*4+4+4*32 -n) {
			case 3:
				cache[1] = make_float4(cache[0].x,DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT);
				break;
			case 2:
				cache[1] = make_float4(cache[0].x,cache[0].y,DEFAULT_FLOAT,DEFAULT_FLOAT);
				break;
			case 1:
				cache[1].w = DEFAULT_FLOAT;
				break;
			default:
				cache[1] = make_float4(DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT);
				break;
			}
		}

		cache[0].x = fmax(cache[0].x,cache[0].y);
		cache[1].x = fmax(cache[1].x,cache[1].y);

		cache[0].z = fmax(cache[0].z,cache[0].w);
		cache[1].z = fmax(cache[1].z,cache[1].w);

		cache[0].x = fmax(cache[0].x,cache[0].z);
		cache[1].x = fmax(cache[1].x,cache[1].z);

		vmax = fmax(vmax,cache[0].x);
		vmax = fmax(vmax,cache[1].x);
		if(vmax == cache[0].x) {
			imax = laneid*4;
			if(vmax == cache[0].y) {
				imax += 1;
			} else if(vmax == cache[0].z) {
				imax += 2;
			} else if(vmax == cache[0].w) {
				imax += 3;
			}

		} else if(vmax == cache[1].x){
			//4*32== the offset of the first half of the fetched number
			// as each thread in the warp fetches a float4
			imax = laneid*4+4*32;
			if(vmax == cache[1].y) {
				imax += 1;
			} else if(vmax == cache[1].z) {
				imax += 2;
			} else if(vmax == cache[1].w) {
				imax += 3;
			}
		}
	} else {
		/*256->512->768->1024
		 *
		 *
		 */
		for(int i = 0; i < n/32*8; i+=2) {
			float4 cache[2];
			cache[0] = ((float4 *) cost)[laneid+32*4*i];
			cache[1] = ((float4 *) cost)[laneid+32*4*(i+1)];
			cache[0].x = fmax(cache[0].x,cache[0].y);
			cache[1].x = fmax(cache[1].x,cache[1].y);

			cache[0].z = fmax(cache[0].z,cache[0].w);
			cache[1].z = fmax(cache[1].z,cache[1].w);

			cache[0].x = fmax(cache[0].x,cache[0].z);
			cache[1].x = fmax(cache[1].x,cache[1].z);
			vmax = fmax(vmax,cache[0].x);
			vmax = fmax(vmax,cache[1].x);

			int index0 = -1;
			index0 += (vmax == cache[0].x);
			index0 += (vmax == cache[0].z)*2;
			index0 += (vmax == cache[0].w);
			index0 += !(vmax == cache[0].z)*(vmax == cache[0].y);
		}
	}
	float t = vmax;
	for (int mask = warpSize/2; mask > 0; mask /= 2)
		vmax = fmax(vmax,__shfl_xor(vmax, mask));

	imax = __ffs(__ballot(vmax == t))-1;

	return imax;
}
#define DEFAULT_FLOAT (0.0f)
template<int lim>
__device__ int __forceinline__ find_entering_var(float * cost, int n) {
	const unsigned int tid = threadIdx.x;
	const unsigned int laneid = tid%32;

	float vmax = DEFAULT_FLOAT;
	int imax = -1;
	/*
	 * cache[0]. x y z w
	 *           0 1 2 3
	 * cache[1]. x y z w
	 *           128 129 130 131
	 */
	if(lim <= 32) {
		if(laneid < n) {
			vmax = cost[laneid];
			if(vmax < 0.0f) {
				imax = laneid;
			}

		}
	} else if(lim <= 64) {
		float2 cache = make_float2(DEFAULT_FLOAT,DEFAULT_FLOAT);
		imax = laneid*2;
		if(laneid*2 < n) {
			cache = ((float2 *) cost)[laneid];
			cache.y *= (laneid*2+1 < n);

		}
		vmax = fmin(cache.x,cache.y);
		if(cache.y == vmax) {
			imax++;
		}
	} else if (lim <= 128) {
		float4 cache = make_float4(DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT);

		if(laneid*4 < n) {
			cache = ((float4 *) cost)[laneid];
			cache.y *= (laneid*4+1 < n);
			cache.z *= (laneid*4+2 < n);
			cache.w *= (laneid*4+3 < n);
		}
		cache.x = fmin(cache.x,cache.y);
		cache.z = fmin(cache.z,cache.w);
		vmax = fmin(cache.x,cache.z);
		//set base index to x
		imax = laneid*4;
		if (vmax == cache.y) {
			imax +=1;
		} else if(vmax == cache.z) {
			imax += 2;
		} else if (vmax == cache.w) {
			imax += 3;
		}
	} else {
		float4 cache = make_float4(DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT);
		float4 cache2 = make_float4(DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT);
		int i;
#pragma unroll 2
		for(i = laneid; i*4+4 < n; i+=32) {

			/* i 0: n 1: 0+4-1= 3: yzw
			 * i 0: n 2: 0+4-2= 2: zw
			 * i 0: n 3: 0+4-3= 1: w
			 * i 0: n 4: 0+4-4= 0:
			 * i 1: n 5: 4+4-5= 3: yzw
			 * i 4: n 4: 16+4-4= 16: xyzw
			 * i 3: n 4: 3+4-4= 3: yzw
			 */
			cache = ((float4 *) cost)[i];
			const int tindex = i*4;
			cache.x = fmin(cache.x,cache.y);
			cache.z = fmin(cache.z,cache.w);
			float tmax = fmin(cache.x,cache.z);
			if(tmax < vmax) {
				vmax = tmax;
				imax = (tmax == cache.x)*(tindex);
				imax = (tmax == cache.y)*(tindex+1);
				imax = (tmax == cache.z)*(tindex+2);
				imax = (tmax == cache.w)*(tindex+3);
			}

		}

		if(i*4 < n) {
			cache = ((float4 *) cost)[i];
			switch(i*4+4 -n) {
			case 3:
				cache = make_float4(cache.x,DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT);
				break;
			case 2:
				cache = make_float4(cache.x,cache.y,DEFAULT_FLOAT,DEFAULT_FLOAT);
				break;
			case 1:
				cache.w = DEFAULT_FLOAT;
				break;
			default:
				cache = make_float4(DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT,DEFAULT_FLOAT);
				break;
			}
			const int tindex = i*4;
			cache.x = fmin(cache.x,cache.y);
			cache.z = fmin(cache.z,cache.w);
			vmax = fmin(cache.x,cache.z);
			if (vmax == cache.y) {
				imax =tindex+1;
			} else if (vmax == cache.w) {
				imax =tindex+3;
			} else if(vmax == cache.z) {
				imax =tindex+2;
			} else if (vmax == cache.x) {
				imax =tindex;
			}
		}

	}

	float t = vmax;
	for (int mask = warpSize/2; mask > 0; mask /= 2)
		vmax = fmin(vmax,__shfl_xor(vmax, mask));

	imax = __shfl(imax,__ffs(__ballot(vmax == t))-1);

	return imax;
}


template<int rows>
__device__ int __forceinline__ get_pivot_row(float  ( *p)[rows],int collumn, int collumns)
{
	const unsigned int laneid = threadIdx.x%32;
	float vmax = INFINITY;
	int index = -1;

	for(int i = laneid; i < rows; i +=32) {
		float div = p[collumn][i];
		float top = p[collumns-1][i];
		if(div > 0.0f) {
			float frac = top/div;
			if(frac < vmax) {
				vmax = frac;
				index = i;
			}
		}
	}
	float t = vmax;
	vmax = fmin(vmax,__shfl_xor(vmax,16));
	vmax = fmin(vmax,__shfl_xor(vmax, 8));
	vmax = fmin(vmax,__shfl_xor(vmax, 4));
	vmax = fmin(vmax,__shfl_xor(vmax, 2));
	vmax = fmin(vmax,__shfl_xor(vmax, 1));
	index = __shfl(index,__ffs(__ballot(vmax == t))-1);
	return index;
}
template<int rows>
__device__ int __forceinline__ is_integer(float  (*p)[rows],int collumns) {
	const unsigned int laneid = threadIdx.x%32;
	float in = p[collumns-1][laneid];
	int result = __all(ceilf(in) == in);
	return result;
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
//only works for 32 rows!!
template<int rows, int lim>
__device__ float __forceinline__ apply_row_op(float  (*p)[rows],float * cost, int collumns)
{

	//
	__shared__ int bids[32][32];
	const unsigned int laneid = threadIdx.x%32;
	const unsigned int warpid = threadIdx.x/32;
	int row, collumn;
	bids[warpid][laneid] = laneid;
	//assert(cost[0] == -2.0f);
	while(1) {
		collumn = find_entering_var<lim>(cost,collumns);
		if(collumn == -1 || cost[collumn] >= 0.0f) {
			break;						
		}

		row = get_pivot_row<rows>(p,collumn,collumns);
		//__syncthreads();
		//if(collumn != 4)
		//printf("col %d row %d, tid %d\n",collumn,row,threadIdx.x);
		//return 0;


		float element = p[collumn][laneid];
		float pivot = __shfl(element,row);
		float y = (-element);
		float costy = -cost[collumn];
		if(laneid == row) {
			cost[collumn] = costy + cost[collumn];
			y = 1.f/element-1.f;
			element /= element;

			bids[warpid][row] = collumn;

		} else {
			element = y*element+element;
		}
		p[collumn][laneid] = element;
		// 0 = xdiv + colpiv
		// -colpiv = xdiv
		// -colpiv/div = x
		for(int c = 0; c < collumn; c++) {
			float element = p[c][laneid];
			float xpp = __shfl(element,row)/pivot;
			element= y*xpp+element;
			p[c][laneid] = element;
			if(laneid == row) {
				cost[c] = costy*element + cost[c];
			//	printf("collumn %d\n",collumn);
			}
		}

		for(int c = collumn+1; c < collumns; c++) {
			float element = p[c][laneid];
			float xpp = __shfl(element,row)/pivot;
			element= y*xpp+element;
			p[c][laneid] = element;
			if(laneid == row) {
				cost[c] = costy*element + cost[c];
			}
		}



	}
//	__syncthreads();
	if(laneid == 0) {
		//printf("tid %d\t", threadIdx.x);
			//printf("%d\t%d\t%d\t%d\t%d\n",bids[warpid][0],bids[warpid][1],bids[warpid][2],bids[warpid][3],bids[warpid][4]);

	}
	assert(is_integer<rows>(p,collumns) == 1);
	return cost[collumns-1];
}
template<int rows>
__device__ int __forceinline__  init_table(float  (*matrix)[rows],float * cost,int n, unsigned int * in, unsigned int * in_value) {
	const unsigned int tid = threadIdx.x;
	const unsigned int laneid = tid % 32;
	const unsigned int mask =  1 << laneid;
	int i;
	for(i = 0; i < n; i ++) {
		matrix[i][laneid] =(float) ((in[i]  & mask) > 0);
	}
	//now i == n
	for(;i<n+32;i++) {
		matrix[i][laneid] = ((float)((i-n) == laneid));
	}
	//now i == n+32, put p collumn
	matrix[i][laneid] = 0.0f;
	i++;
	//put constraint
	matrix[i][laneid] = 1.0f;

	int c;
	for(c = laneid; c< n; c +=32) {
		cost[c] = -((float) in_value[c]);
	}
	for(;c < n+32; c += 32) {
		cost[c] = 0.0f;
	}
	// now c = n+32
	if(laneid == 0) {
		cost[n+32] = 1.0f;
		cost[n+32+1] = 0.0f;
	}
	return i+1;
}


template<int rows>
__global__ void do_simplex(float  (*matrix)[rows],float * cost,int n,unsigned int * in, unsigned int * in_value) {


	const unsigned int tid = threadIdx.x;
	const unsigned int warpid = (threadIdx.x+ blockIdx.x * blockDim.x) / 32;
	const unsigned int laneid = tid % 32;

	__shared__ int cache2[1024];
	int new_n = init_table<rows>(matrix,cost,n,in,in_value);
	cache2[tid] = apply_row_op<32,60>((matrix+warpid*n),(cost+warpid*n),new_n);
	//__threadfence_system();
	__syncthreads();
	//assert(collumn == cache2[tid]);
	//printf("t1 %d i %d, warp %d\n",cache2[tid],tid,warpid);
	for(int i = 0; i < 1024;i++) {

		assert(cache2[i] == cache2[tid]);
	}
	//assert(cache[laneid][laneid] == cache[warpid][laneid]);

}

#define set(X) (1<<(X-1))
#define set2(X,Y) (set(X)|set(Y))
int main(int argc, const char* argv[]) {
	int warps = 1024*14/32;
	const int problemwidth = 21;
	const int rows = 32;
	const int collumns = problemwidth+2+rows;
	float matrix[collumns][rows];
	unsigned int bids[problemwidth] = {set(5),set2(4,5),set2(2,4),set2(2,3),set(3),set2(1,3),
										46554,88465,122321,4654848,6545645,1321321,3215484,64555,
										665565,12324,32132,32122,548484,989498,456542};
	unsigned int value [problemwidth] = {2,3,4,6,8,1,14,58,64,21,32,45,65,1,23,45,84,65,32,12,45};
	float cost[collumns];
	//+1 for the constraints
	int i;
	for(i = 0; i < problemwidth;i++) {
		for(int j = 0; j < rows;j++) {
			matrix[i][j] = 0.0 + ((float) !!(set(j) & bids[i]));

		}
		cost[i] =-((float) value[i]);
		//printf("%f\n",cost[i]);
	}
	//return 0;
	//put the I matrix in mem
	for(; i < problemwidth+rows;i++) {
		for(int j =0; j < rows;j++) {
			matrix[i][j] = 0.0f;
			if(j == i-problemwidth) {
				matrix[i][j] = 1.0f;
			}
		}
	}

	//set the constraints
	for(int j=0; j < rows;j++) {
		matrix[problemwidth+rows][j] = 0.0f;
		matrix[problemwidth+rows+1][j] = 1.0f;
	}

	for(i = 0; i < rows; i ++ ) {
		for(int c = 0; c < collumns; c++) {
			printf("%.1f\t",matrix[c][i]);
		}
		printf("\n");
	}
	//set rest of cost vector to 0
	for(int c = problemwidth; c < collumns; c++) {
		cost[c] = 0.0f;
	}
	//set p to 1
	cost[collumns -2] = 1.0f;

	for(int c = 0; c < collumns; c++) {
		printf("%.1f\t",cost[c]);
	}

	float (* dmatrix)[32];
	CUDA_CHECK_RETURN(hipMalloc((void**) &dmatrix,sizeof(matrix)*warps));
	printf("allocated %lu bytes for matrix\n",sizeof(matrix)*warps);
	float * mmatrix = (float *)  dmatrix;
	float * dcost;
	unsigned int * in_value;
	unsigned int * in;
	CUDA_CHECK_RETURN(hipMalloc((void**) &in,sizeof(unsigned int)*problemwidth));
	CUDA_CHECK_RETURN(hipMalloc((void**) &in_value,sizeof(unsigned int)*problemwidth));
	CUDA_CHECK_RETURN(hipMalloc((void**) &dcost,sizeof(cost)*warps));
	printf("allocated %lu bytes for cost\n",sizeof(cost)*warps);

	CUDA_CHECK_RETURN(hipMemcpy(in,bids,sizeof(unsigned int)*problemwidth, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(in_value,value,sizeof(unsigned int)*problemwidth, hipMemcpyHostToDevice));
	//return 0;
//	for(int w = 0; w < warps; w++) {
	//	CUDA_CHECK_RETURN(hipMemcpy(&mmatrix[w*(sizeof(matrix)/sizeof(float))],matrix,sizeof(matrix), hipMemcpyHostToDevice));
	//	CUDA_CHECK_RETURN(hipMemcpy(&dcost[w*(sizeof(cost)/sizeof(float))],cost,sizeof(cost), hipMemcpyHostToDevice));
	//}

	//CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(float) * 26843545));
	printf("hello\n");
	do_simplex<32><<<14,1024>>>(dmatrix,dcost,problemwidth,in,in_value);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//return 0;
	CUDA_CHECK_RETURN(hipMemcpy(matrix,mmatrix,sizeof(matrix), hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(cost,dcost,sizeof(cost), hipMemcpyDeviceToHost));
	for(i = 0; i < rows; i ++ ) {
			for(int c = 0; c < collumns; c++) {
				printf("%.1f\t",matrix[c][i]);
			}
			printf("\n");
		}
	for(int c = 0; c < collumns; c++) {
		printf("%.1f\t",cost[c]);
	}
	exit(0);
	if(argc < 2) {
		fprintf(stderr,"No argument supplied\n");
		exit(EXIT_FAILURE);
	}
	for(int i = 0; i < argc; i++) {
		printf("Argument %d : %s\n",i,argv[i]);
	}

	struct config * config = parse_file(argv[1]);
	//	void *d = NULL;
	//	int i;
	//	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];
	//
	//	for (i = 0; i < WORK_SIZE; i++)
	//		idata[i] = (unsigned int) i;
	//
	//	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	//	CUDA_CHECK_RETURN(
	//			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));
	//
	//	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);
	//
	//	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	//	CUDA_CHECK_RETURN(hipGetLastError());
	//	CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));
	//
	//	for (i = 0; i < WORK_SIZE; i++)
	//		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);
	//
	//	CUDA_CHECK_RETURN(hipFree((void*) d));
	//	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}
